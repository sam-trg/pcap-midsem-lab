#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void reverseWords(char *input, char *output, int wordLength) {
    int idx = threadIdx.x;
    int wordStart = idx * (wordLength + 1); // +1 accounts for spaces

    for (int i = 0; i < wordLength; i++) {
        output[wordStart + i] = input[wordStart + (wordLength - i - 1)];
    }

    // Copy space if it exists
    if (input[wordStart + wordLength] == ' ') {
        output[wordStart + wordLength] = ' ';
    }
}

int main() {
    const char input[] = "cuda fast test word";
    const int wordLength = 4;
    char output[sizeof(input)] = "";

    char *d_input, *d_output;
    hipMalloc(&d_input, sizeof(input));
    hipMalloc(&d_output, sizeof(output));

    hipMemcpy(d_input, input, sizeof(input), hipMemcpyHostToDevice);

    reverseWords<<<1, 4>>>(d_input, d_output, wordLength);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, sizeof(output), hipMemcpyDeviceToHost);

    printf("Reversed Words: %s\n", output);

    hipFree(d_input); 
    hipFree(d_output);
    return 0;
}
