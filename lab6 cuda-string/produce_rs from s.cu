#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void generateRS(char *S, char *RS, int len) {
    int idx = threadIdx.x;
    RS[2 * idx] = S[idx];
    RS[2 * idx + 1] = S[idx];
}

int main() {
    const char S[] = "PCAP";
    char RS[9] = "";

    char *d_S, *d_RS;
    hipMalloc(&d_S, sizeof(S));
    hipMalloc(&d_RS, sizeof(RS));

    hipMemcpy(d_S, S, sizeof(S), hipMemcpyHostToDevice);

    generateRS<<<1, 4>>>(d_S, d_RS, 4);

    hipMemcpy(RS, d_RS, sizeof(RS), hipMemcpyDeviceToHost);

    printf("Output String RS: %s\n", RS);

    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}
