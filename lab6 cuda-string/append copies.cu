#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void createStringT(char *Sin, char *T, int N) {
    int idx = threadIdx.x;
    for (int i = 0; i < N; i++) {
        T[idx * N + i] = Sin[idx];
    }
}

int main() {
    const char Sin[] = "Hai";
    const int N = 3;
    char T[10] = "";

    char *d_Sin, *d_T;
    hipMalloc(&d_Sin, sizeof(Sin));
    hipMalloc(&d_T, sizeof(T));

    hipMemcpy(d_Sin, Sin, sizeof(Sin), hipMemcpyHostToDevice);

    createStringT<<<1, 3>>>(d_Sin, d_T, N);

    hipMemcpy(T, d_T, sizeof(T), hipMemcpyDeviceToHost);

    printf("Output String T: %s\n", T);

    hipFree(d_Sin);
    hipFree(d_T);

    return 0;
}
