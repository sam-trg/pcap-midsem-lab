#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void repeatString(char *Sin, char *Sout, int len, int N) {
    int idx = threadIdx.x;
    for (int i = 0; i < N; i++) {
        Sout[idx * N + i] = Sin[idx];
    }
}

int main() {
    const char Sin[] = "Hello";
    const int N = 3;
    char Sout[16] = "";

    char *d_Sin, *d_Sout;
    hipMalloc(&d_Sin, sizeof(Sin));
    hipMalloc(&d_Sout, sizeof(Sout));

    hipMemcpy(d_Sin, Sin, sizeof(Sin), hipMemcpyHostToDevice);

    repeatString<<<1, 5>>>(d_Sin, d_Sout, 5, N);

    hipMemcpy(Sout, d_Sout, sizeof(Sout), hipMemcpyDeviceToHost);

    printf("Repeated String Sout: %s\n", Sout);

    hipFree(d_Sin);
    hipFree(d_Sout);

    return 0;
}
