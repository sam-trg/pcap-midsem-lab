#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void countWord(char *text, char *word, int *count, int textLen, int wordLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx <= textLen - wordLen) {
        bool match = true;
        for (int i = 0; i < wordLen; i++) {
            if (text[idx + i] != word[i]) {
                match = false;
                break;
            }
        }
        if (match) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    const char text[] = "cuda is fast cuda is powerful cuda cuda";
    const char word[] = "cuda";
    int count = 0;

    char *d_text, *d_word;
    int *d_count;

    hipMalloc(&d_text, sizeof(text));
    hipMalloc(&d_word, sizeof(word));
    hipMalloc(&d_count, sizeof(int));

    hipMemcpy(d_text, text, sizeof(text), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, sizeof(word), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    countWord<<<1, strlen(text)>>>(d_text, d_word, d_count, strlen(text), strlen(word));

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("Count of word '%s': %d\n", word, count);

    hipFree(d_text); hipFree(d_word); hipFree(d_count);
    return 0;
}
