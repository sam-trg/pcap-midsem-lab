#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void linearAlgebra(float *x, float *y, float alpha, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        y[idx] = alpha * x[idx] + y[idx];
}

int main() {
    const int N = 10;
    float x[N], y[N];
    float alpha = 2.0f;

    for (int i = 0; i < N; i++) {
        x[i] = i * 1.0f;
        y[i] = i * 0.5f;
    }

    float *d_x, *d_y;
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    linearAlgebra<<<1, N>>>(d_x, d_y, alpha, N);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Resultant y values: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", y[i]);
    }
    printf("\n");
    hipFree(d_x); hipFree(d_y);
    return 0;
}
