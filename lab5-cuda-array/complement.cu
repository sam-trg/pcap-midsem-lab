#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void onesComplement(int *input, int *output, int N) {
    int idx = threadIdx.x;
    if (idx < N)
        output[idx] = ~input[idx];
}

int main() {
    const int N = 5;
    int input[N] = {0b1010, 0b1100, 0b0011, 0b0101, 0b1001};
    int output[N];

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, N * sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);

    onesComplement<<<1, N>>>(d_input, d_output, N);

    hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("One's Complement: ");
    for (int i = 0; i < N; i++)
        printf("%d ", output[i]);

    hipFree(d_input); hipFree(d_output);
    return 0;
}
