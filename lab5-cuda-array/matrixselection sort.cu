#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void selectionSort(int *matrix, int N) {
    int row = blockIdx.x;
    for (int i = 0; i < N - 1; i++) {
        int minIdx = i;
        for (int j = i + 1; j < N; j++) {
            if (matrix[row * N + j] < matrix[row * N + minIdx])
                minIdx = j;
        }
        int temp = matrix[row * N + i];
        matrix[row * N + i] = matrix[row * N + minIdx];
        matrix[row * N + minIdx] = temp;
    }
}

int main() {
    const int N = 4;
    int matrix[N][N] = {{3, 2, 1, 4}, {6, 5, 4, 3}, {9, 8, 7, 6}, {12, 11, 10, 9}};
    int *d_matrix;

    hipMalloc((void**)&d_matrix, N * N * sizeof(int));
    hipMemcpy(d_matrix, matrix, N * N * sizeof(int), hipMemcpyHostToDevice);

    selectionSort<<<N, 1>>>(d_matrix, N);

    hipMemcpy(matrix, d_matrix, N * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted Matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            printf("%d ", matrix[i][j]);
        printf("\n");
    }

    hipFree(d_matrix);
    return 0;
}
