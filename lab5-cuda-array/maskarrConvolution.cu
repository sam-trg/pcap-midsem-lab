#include <stdio.h>
#include <hip/hip_runtime.h>

#define MASK_WIDTH 3

__global__ void convolution1D(int *N, int *M, int *P, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int radius = MASK_WIDTH / 2;

    if (idx < width) {
        int result = 0;
        for (int i = -radius; i <= radius; i++) {
            int neighbor_idx = idx + i;
            if (neighbor_idx >= 0 && neighbor_idx < width) {
                result += N[neighbor_idx] * M[radius + i];
            }
        }
        P[idx] = result;
    }
}

int main() {
    const int width = 10;
    int N[width] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int M[MASK_WIDTH] = {1, 0, -1};
    int P[width];

    int *d_N, *d_M, *d_P;
    hipMalloc(&d_N, width * sizeof(int));
    hipMalloc(&d_M, MASK_WIDTH * sizeof(int));
    hipMalloc(&d_P, width * sizeof(int));

    hipMemcpy(d_N, N, width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, MASK_WIDTH * sizeof(int), hipMemcpyHostToDevice);

    convolution1D<<<(width + 255) / 256, 256>>>(d_N, d_M, d_P, width);

    hipMemcpy(P, d_P, width * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant Array: ");
    for (int i = 0; i < width; i++)
        printf("%d ", P[i]);
    
    printf("\n");

    hipFree(d_N); hipFree(d_M); hipFree(d_P);
    return 0;
}
