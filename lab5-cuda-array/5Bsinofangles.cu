#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void computeSine(float *angles, float *sines, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        sines[idx] = sinf(angles[idx]);
}

int main() {
    const int N = 10;
    float angles[N], sines[N];
    float *d_angles, *d_sines;

    for (int i = 0; i < N; i++) angles[i] = i * 0.1f;

    hipMalloc((void**)&d_angles, N * sizeof(float));
    hipMalloc((void**)&d_sines, N * sizeof(float));

    hipMemcpy(d_angles, angles, N * sizeof(float), hipMemcpyHostToDevice);

    computeSine<<<1, N>>>(d_angles, d_sines, N);

    hipMemcpy(sines, d_sines, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Sine values: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", sines[i]);
    }
    printf("\n");
    hipFree(d_angles); hipFree(d_sines);
    return 0;
}
