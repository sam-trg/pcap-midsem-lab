#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void convertToOctal(int *input, int *output, int N) {
    int idx = threadIdx.x;
    if (idx < N) {
        int num = input[idx];
        int octal = 0, place = 1;
        while (num > 0) {
            octal += (num % 8) * place;
            num /= 8;
            place *= 10;
        }
        output[idx] = octal;
    }
}

int main() {
    const int N = 5;
    int input[N] = {10, 20, 30, 40, 50};
    int output[N];

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, N * sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);

    convertToOctal<<<1, N>>>(d_input, d_output, N);

    hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Octal values: ");
    for (int i = 0; i < N; i++)
        printf("%d ", output[i]);

    printf("\n");
    
    hipFree(d_input); hipFree(d_output);
    return 0;
}
