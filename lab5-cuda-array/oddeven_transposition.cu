#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void oddEvenSort(int *arr, int N) {
    int idx = threadIdx.x;
    for (int i = 0; i < N; i++) {
        if ((i % 2 == 0 && idx % 2 == 0) || (i % 2 != 0 && idx % 2 != 0)) {
            if (idx < N - 1 && arr[idx] > arr[idx + 1]) {
                int temp = arr[idx];
                arr[idx] = arr[idx + 1];
                arr[idx + 1] = temp;
            }
        }
        __syncthreads();
    }
}

int main() {
    const int N = 10;
    int arr[N] = {10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
    int *d_arr;

    hipMalloc(&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);

    oddEvenSort<<<1, N>>>(d_arr, N);

    hipMemcpy(arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted Array: ");
    for (int i = 0; i < N; i++)
        printf("%d ", arr[i]);

    printf("\n");
    
    hipFree(d_arr);
    return 0;
}
