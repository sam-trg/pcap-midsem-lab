#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int idx = threadIdx.x;  // Each thread handles one element
    if (idx < N)
        c[idx] = a[idx] + b[idx];
}

int main() {
    const int N = 10;
    int a[N], b[N], c[N];
    int *d_a, *d_b, *d_c;

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<1, N>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }

    printf("\n");

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
